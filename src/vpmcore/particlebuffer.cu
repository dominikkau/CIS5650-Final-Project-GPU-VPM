#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "particlebuffer.h"

// Allocates memory for fields in bufferMask
// Ignores fields that have been allocated already
void ParticleBuffer::mallocFields(unsigned int numParticles, int bufferMask) {
    // Ignore fields that have been allocated
    bufferMask &= ~bufferFields;

    switch (bufferType) {
    case BUFFER_DEVICE:
        if (bufferMask & BUFFER_X) {
            hipMalloc((void**)&X, numParticles * sizeof(vpmvec3));
            checkCUDAError("hipMalloc of X failed!");
        }

        if (bufferMask & BUFFER_U) {
            hipMalloc((void**)&U, numParticles * sizeof(vpmvec3));
            checkCUDAError("hipMalloc of U failed!");
        }

        if (bufferMask & BUFFER_J) {
            hipMalloc((void**)&J, numParticles * sizeof(vpmmat3));
            checkCUDAError("hipMalloc of J failed!");
        }

        if (bufferMask & BUFFER_GAMMA) {
            hipMalloc((void**)&Gamma, numParticles * sizeof(vpmvec3));
            checkCUDAError("hipMalloc of Gamma failed!");
        }

        if (bufferMask & BUFFER_SIGMA) {
            hipMalloc((void**)&sigma, numParticles * sizeof(vpmfloat));
            checkCUDAError("hipMalloc of sigma failed!");
        }

        if (bufferMask & BUFFER_SFS) {
            hipMalloc((void**)&SFS, numParticles * sizeof(vpmvec3));
            checkCUDAError("hipMalloc of SFS failed!");
        }

        if (bufferMask & BUFFER_C) {
            hipMalloc((void**)&C, numParticles * sizeof(vpmvec3));
            checkCUDAError("hipMalloc of C failed!");
        }

        if (bufferMask & BUFFER_M) {
            hipMalloc((void**)&M, numParticles * sizeof(vpmmat3));
            checkCUDAError("hipMalloc of M failed!");
        }

        if (bufferMask & BUFFER_INDEX) {
            hipMalloc((void**)&index, numParticles * sizeof(int));
            checkCUDAError("hipMalloc of index failed!");
        }

        /*hipMalloc((void**)&PSE, size * sizeof(vpmvec3));
        checkCUDAError("hipMalloc of PSE failed!");

        hipMalloc((void**)&isStatic, size * sizeof(bool));
        checkCUDAError("hipMalloc of isStatic failed!");

        hipMalloc((void**)&vol, size * sizeof(vpmfloat));
        checkCUDAError("hipMalloc of vol failed!");

        hipMalloc((void**)&circulation, size * sizeof(vpmfloat));
        checkCUDAError("hipMalloc of circulation failed!");*/
        break;

    case BUFFER_HOST:
        if (bufferMask & BUFFER_X) X = new vpmvec3[numParticles];
        if (bufferMask & BUFFER_U) U = new vpmvec3[numParticles];
        if (bufferMask & BUFFER_J) J = new vpmmat3[numParticles];
        if (bufferMask & BUFFER_GAMMA) Gamma = new vpmvec3[numParticles];
        if (bufferMask & BUFFER_SIGMA) sigma = new vpmfloat[numParticles];
        if (bufferMask & BUFFER_SFS) SFS = new vpmvec3[numParticles];
        if (bufferMask & BUFFER_C) C = new vpmvec3[numParticles];
        if (bufferMask & BUFFER_M) M = new vpmmat3[numParticles];
        if (bufferMask & BUFFER_INDEX) index = new int[numParticles];
        /*if (bufferMask & BUFFER_PSE) PSE = new vpmvec3[numParticles];
        if (bufferMask & BUFFER_IS_STATIC) isStatic = new bool[numParticles];
        if (bufferMask & BUFFER_VOL) vol = new vpmfloat[numParticles];
        if (bufferMask & BUFFER_CIRC) circulation = new vpmfloat[numParticles];*/
        break;

    case BUFFER_HOST_PINNED:
        if (bufferMask & BUFFER_X) hipHostMalloc((void**)&X, numParticles * sizeof(vpmvec3));
        if (bufferMask & BUFFER_U) hipHostMalloc((void**)&U, numParticles * sizeof(vpmvec3));
        if (bufferMask & BUFFER_J) hipHostMalloc((void**)&J, numParticles * sizeof(vpmmat3));
        if (bufferMask & BUFFER_GAMMA) hipHostMalloc((void**)&Gamma, numParticles * sizeof(vpmvec3));
        if (bufferMask & BUFFER_SIGMA) hipHostMalloc((void**)&sigma, numParticles * sizeof(vpmfloat));
        if (bufferMask & BUFFER_SFS) hipHostMalloc((void**)&SFS, numParticles * sizeof(vpmvec3));
        if (bufferMask & BUFFER_C) hipHostMalloc((void**)&C, numParticles * sizeof(vpmvec3));
        if (bufferMask & BUFFER_M) hipHostMalloc((void**)&M, numParticles * sizeof(vpmmat3));
        if (bufferMask & BUFFER_INDEX) hipHostMalloc((void**)&index, numParticles * sizeof(int));
        /*if (bufferMask & BUFFER_PSE) hipHostMalloc((void**)&PSE, numParticles * sizeof(vpmvec3));
        if (bufferMask & BUFFER_IS_STATIC) hipHostMalloc((void**)&isStatic, numParticles * sizeof(bool));
        if (bufferMask & BUFFER_VOL) hipHostMalloc((void**)&vol, numParticles * sizeof(vpmfloat));
        if (bufferMask & BUFFER_CIRC) hipHostMalloc((void**)&circulation, numParticles * sizeof(vpmfloat));*/
        break;
    }

    // Update allocated fields
    bufferFields |= bufferMask;
}

// Frees memory for fields in bufferMask
// Ignores fields that have not been allocated
void ParticleBuffer::freeFields(int bufferMask) {
    // Ignore unallocated fields
    bufferMask &= bufferFields;

    switch (bufferType) {
    case BUFFER_DEVICE:
        if (bufferMask & BUFFER_X) hipFree(X);
        if (bufferMask & BUFFER_U) hipFree(U);
        if (bufferMask & BUFFER_J) hipFree(J);
        if (bufferMask & BUFFER_GAMMA) hipFree(Gamma);
        if (bufferMask & BUFFER_SIGMA) hipFree(sigma);
        if (bufferMask & BUFFER_SFS) hipFree(SFS);
        if (bufferMask & BUFFER_C) hipFree(C);
        if (bufferMask & BUFFER_M) hipFree(M);
        if (bufferMask & BUFFER_INDEX) hipFree(index);
        /*if (bufferMask & BUFFER_PSE) hipFree(PSE);
        if (bufferMask & BUFFER_IS_STATIC) hipFree(isStatic);
        if (bufferMask & BUFFER_VOL) hipFree(vol);
        if (bufferMask & BUFFER_CIRC) hipFree(circulation);*/
        break;

    case BUFFER_HOST:
        if (bufferMask & BUFFER_X) delete[] X;
        if (bufferMask & BUFFER_U) delete[] U;
        if (bufferMask & BUFFER_J) delete[] J;
        if (bufferMask & BUFFER_GAMMA) delete[] Gamma;
        if (bufferMask & BUFFER_SIGMA) delete[] sigma;
        if (bufferMask & BUFFER_SFS) delete[] SFS;
        if (bufferMask & BUFFER_C) delete[] C;
        if (bufferMask & BUFFER_M) delete[] M;
        if (bufferMask & BUFFER_INDEX) delete[] index;
        /*if (bufferMask & BUFFER_PSE) delete[] PSE;
        if (bufferMask & BUFFER_IS_STATIC) delete[] isStatic;
        if (bufferMask & BUFFER_VOL) delete[] vol;
        if (bufferMask & BUFFER_CIRC) delete[] circulation;*/
        break;

    case BUFFER_HOST_PINNED:
        if (bufferMask & BUFFER_X) hipHostFree(X);
        if (bufferMask & BUFFER_U) hipHostFree(U);
        if (bufferMask & BUFFER_J) hipHostFree(J);
        if (bufferMask & BUFFER_GAMMA) hipHostFree(Gamma);
        if (bufferMask & BUFFER_SIGMA) hipHostFree(sigma);
        if (bufferMask & BUFFER_SFS) hipHostFree(SFS);
        if (bufferMask & BUFFER_C) hipHostFree(C);
        if (bufferMask & BUFFER_M) hipHostFree(M);
        if (bufferMask & BUFFER_INDEX) hipHostFree(index);
        /*if (bufferMask & BUFFER_PSE) hipHostFree(PSE);
        if (bufferMask & BUFFER_IS_STATIC) hipHostFree(isStatic);
        if (bufferMask & BUFFER_VOL) hipHostFree(vol);
        if (bufferMask & BUFFER_CIRC) hipHostFree(circulation);*/
        break;
    }

    // Update allocated fields
    bufferFields &= ~bufferMask;
}

void _cpyParticleBuffer(ParticleBuffer destBuffer, ParticleBuffer srcBuffer,
    unsigned int destIndex, unsigned int srcNumParticles, int bufferMask, hipStream_t stream) {

    // Determine hipMemcpy direction
    hipMemcpyKind cpyDirection;
    if (destBuffer.bufferType == BUFFER_DEVICE) {
        if (srcBuffer.bufferType == BUFFER_DEVICE) cpyDirection = hipMemcpyDeviceToDevice;
        else cpyDirection = hipMemcpyHostToDevice;
    }
    else {
        if (srcBuffer.bufferType == BUFFER_DEVICE) cpyDirection = hipMemcpyDeviceToHost;
        else cpyDirection = hipMemcpyHostToHost;
    }

    // Ensure that we do not try to copy from or to non-existent fields
    bufferMask &= (destBuffer.bufferFields & srcBuffer.bufferFields);

    if (bufferMask & BUFFER_X) {
        hipMemcpyAsync(destBuffer.X + destIndex, srcBuffer.X, srcNumParticles * sizeof(vpmvec3), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_U) {
        hipMemcpyAsync(destBuffer.U + destIndex, srcBuffer.U, srcNumParticles * sizeof(vpmvec3), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_J) {
        hipMemcpyAsync(destBuffer.J + destIndex, srcBuffer.J, srcNumParticles * sizeof(vpmmat3), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_GAMMA) {
        hipMemcpyAsync(destBuffer.Gamma + destIndex, srcBuffer.Gamma, srcNumParticles * sizeof(vpmvec3), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_SIGMA) {
        hipMemcpyAsync(destBuffer.sigma + destIndex, srcBuffer.sigma, srcNumParticles * sizeof(vpmfloat), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_SFS) {
        hipMemcpyAsync(destBuffer.SFS + destIndex, srcBuffer.SFS, srcNumParticles * sizeof(vpmvec3), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_C) {
        hipMemcpyAsync(destBuffer.C + destIndex, srcBuffer.C, srcNumParticles * sizeof(vpmvec3), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_M) {
        hipMemcpyAsync(destBuffer.M + destIndex, srcBuffer.M, srcNumParticles * sizeof(vpmmat3), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_INDEX) {
        hipMemcpyAsync(destBuffer.index + destIndex, srcBuffer.index, srcNumParticles * sizeof(int), cpyDirection, stream);
    }
    /*if (bufferMask & BUFFER_PSE) {
        hipMemcpyAsync(destBuffer.PSE + destIndex, srcBuffer.PSE, srcNumParticles * sizeof(vpmvec3), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_IS_STATIC) {
        hipMemcpyAsync(destBuffer.isStatic + destIndex, srcBuffer.isStatic, srcNumParticles * sizeof(bool), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_VOL) {
        hipMemcpyAsync(destBuffer.vol + destIndex, srcBuffer.vol, srcNumParticles * sizeof(vpmfloat), cpyDirection, stream);
    }
    if (bufferMask & BUFFER_CIRC) {
        hipMemcpyAsync(destBuffer.circulation + destIndex, srcBuffer.circulation, srcNumParticles * sizeof(vpmfloat), cpyDirection, stream);
    }*/
}

unsigned int cpyParticleBuffer(ParticleBuffer destBuffer, ParticleBuffer srcBuffer, unsigned int destNumParticles,
    unsigned int destMaxParticles, unsigned int srcNumParticles, unsigned int destIndex, int bufferMask, hipStream_t stream) {

    // Start index exceeds maximum number of particles
    if (destIndex >= destMaxParticles) return destNumParticles;

    // Do not leave undefined particles between existing and copied
    if (destIndex > destNumParticles) destIndex = destNumParticles;

    // Number of particles to be copied is limited by destMaxParticles
    srcNumParticles = min(srcNumParticles, destMaxParticles - destIndex);

    _cpyParticleBuffer(destBuffer, srcBuffer, destIndex, srcNumParticles, bufferMask, stream);

    // Calculate new number of particles
    if (destIndex + srcNumParticles >= destNumParticles) {
        destNumParticles = destIndex + srcNumParticles;
    }

    return destNumParticles;
}